#include <hip/hip_runtime.h>

#include <bits/stdc++.h>
#include <chrono>

#define MAX_VAL 4294967295

using namespace std;
void printvec(vector<uint> &a,ofstream &outstr, int m){
	for(int i = 0;i < a.size()/m;i++){
		for(int j=0; j < m; j++)
			outstr << a[i*m + j] << " ";
		outstr << endl;
	}
}

int givint(char *buffer){
	int *res = (int *)(buffer);
	return *res;
}

bool isZero(uint *v, int l){
	bool flg = true;
	for (int i = 0; i < l; ++i)
	{	
		if(v[i] != 0){
			flg = false;
			break;
		}
	}
	return flg;
}

__global__
void matMulGPU(uint *a, uint *b, uint *c, int *ka, int *kb, int *rof, int *col, int* row, int *cof, int m, int n, int k1, int k2){
	extern __shared__ uint dab[];
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int nm = n/m;
	int i = bid / nm;
	int k = bid % nm;
	uint64_t temp = 0;
	int j1 = rof[i];
	int j2 = cof[k];
	while (j1 < rof[i+1] && j2 < cof[k+1]){
		int id1 = ka[j1];
		int id2 = kb[j2];
		int cl = col[j1];
		int rw = row[j2];
		int flag = 0;
		if(cl == rw){
			flag = 1;
			j1++;
			j2++;
		}
		else if(cl < rw)
			j1++;
		else
			j2++;
		// if(tid == 0 && bid == 9)
		// 	printf("%d %d : %d %d : %d %d\n",i,cl,cl,k,id1,id2);
		if(flag){
			__syncthreads();
			dab[tid] = (uint)a[tid + id1*m*m];
			dab[tid + m*m] = (uint)b[tid + id2*m*m];
			__syncthreads();
			int ii = tid/m;
			int jj = tid%m;
			for (int kk = 0; kk < m; ++kk)
			{
				temp = temp + (uint64_t)(dab[ii*m + kk] * dab[kk*m + jj + m*m]);
			}
		}
	}
	c[tid + i*m*n + k*m*m] = min(temp,MAX_VAL);
}

void matMul(vector<array<int,3>> &mp1, vector<uint> &blksA, vector<array<int,3>> &mp2, vector<uint> &blksB,  long long n, long long m, vector<uint> &blksC){
	long long nm = n/m;
	// sending data to GPU
	int streamSize = 2;
	hipError_t err;
	hipStream_t stream[streamSize];
	for(int i = 0;i<streamSize ;i++){
		hipStreamCreate(&stream[i]);
	}
	size_t size = sizeof(uint);
	size_t size2 = sizeof(int);
	size_t size3 = sizeof(uint) * n * n;
	uint *a = &blksA[0], *b = &blksB[0], *da, *db;
	uint *c = &blksC[0], *dc;
	hipMalloc(&da,size*blksA.size());
	hipMalloc(&db,size*blksB.size());
	hipMalloc(&dc,size3);
	hipDeviceSynchronize();
	hipMemset(dc,0,size3);
	hipMemcpyAsync(da,a,(size_t)size*(size_t)blksA.size(),hipMemcpyHostToDevice,stream[0]);
	hipMemcpyAsync(db,b,(size_t)size*(size_t)blksB.size(),hipMemcpyHostToDevice,stream[1]);

	// converting to CSR
	vector<int> valV;
	vector<int> colV;
	vector<int> rofV;
	int offset = 0;
	int rowno = 0;
	rofV.push_back(offset);
	for(int i = 0; i <mp1.size() ; i++){
		colV.push_back(mp1[i][1]);
		valV.push_back(mp1[i][2]);
		if(mp1[i][0] > rowno){
			for(int cc = 0;cc<(mp1[i][0]-rowno);cc++){
				rofV.push_back(offset);
			}
			rowno = mp1[i][0];
		}
		offset+=1;
	}
	for(int j = rowno;j<nm;j++){
		rofV.push_back(blksA.size()/m/m);
	}
	vector<int> valV2;
	vector<int> cofV;
	vector<int> rowV;
	offset = 0;
	int colno = 0;
	cofV.push_back(offset);
	for(int i = 0; i <mp2.size() ; i++){
		rowV.push_back(mp2[i][0]);
		valV2.push_back(mp2[i][2]);
		if(mp2[i][1] > colno){
			for(int cc = 0;cc<(mp2[i][1]-colno);cc++){
				cofV.push_back(offset);
			}
			colno = mp2[i][1];
		}
		offset+=1;
	}
	for(int j = colno;j<nm;j++){
		cofV.push_back(blksB.size()/m/m);
	}
	std::cout << "CSR converted\n";

	int *ka;
	hipMalloc(&ka,(size_t)size2*(size_t)mp1.size());
	hipMemcpyAsync(ka,valV.data(),(size_t)size2*(size_t)mp1.size(),hipMemcpyHostToDevice,stream[0]);
	int *kb;
	hipMalloc(&kb,(size_t)size2*(size_t)mp2.size());
	hipMemcpyAsync(kb,valV2.data(),(size_t)size2*(size_t)mp2.size(),hipMemcpyHostToDevice,stream[1]);
	int *rof, *col, *cof, *row;
	hipMalloc(&rof,(size_t)rofV.size()*size2);
	hipMalloc(&col,(size_t)colV.size()*size2);
	hipMalloc(&cof,(size_t)cofV.size()*size2);
	hipMalloc(&row,(size_t)rowV.size()*size2);
	hipMemcpyAsync(rof,&rofV[0],(size_t)rofV.size()*size2,hipMemcpyHostToDevice,stream[0]);
	hipMemcpyAsync(col,&colV[0],(size_t)colV.size()*size2,hipMemcpyHostToDevice,stream[1]);
	hipMemcpyAsync(cof,&cofV[0],(size_t)cofV.size()*size2,hipMemcpyHostToDevice,stream[0]);
	hipMemcpyAsync(row,&rowV[0],(size_t)rowV.size()*size2,hipMemcpyHostToDevice,stream[1]);

	int stride = (int)(nm*nm);
	// chrono::time_point<std::chrono::system_clock> startg = std::chrono::system_clock::now();
	hipDeviceSynchronize();
	matMulGPU<<<stride,m*m,2*size*m*m,0>>>(da,db,dc,ka,kb,rof,col,row,cof,m,n,mp1.size(),mp2.size()); // i X k
	// cudaDeviceSynchronize();
	// chrono::time_point<std::chrono::system_clock> endg = std::chrono::system_clock::now();
	// chrono::duration<double> elapsed_secondsg = endg-startg;
	// std::cout << "gpu multiplication time: " << elapsed_secondsg.count() << "s\n";

	err = hipMemcpy((void *)c,(void *)dc,size3,hipMemcpyDeviceToHost);
	if (err != hipSuccess) 
	    printf("Error: %s\n", hipGetErrorString(err));

	// free the memory
	hipFree(da);
	hipFree(db);
	hipFree(ka);
	hipFree(kb);
	hipFree(rof);
	hipFree(col);
	hipFree(dc);
	for(int i = 0;i<streamSize ;i++){
		hipStreamDestroy(stream[i]);
	}
}

bool compmy(const array<int,3> &a, const array<int,3> &b){
	if(a[1] == b[1]){
		if(a[0] == b[0]){
			return a[2] < b[2];
		}
		return a[0] < b[0];
	}
	return a[1] < b[1];
};

int main(int argc, char const *argv[])
{
	string inputA = "";
	string inputB = "";
	string outputC = "";

	// int thrds = 8;
	// std::ofstream myFile;
	// myFile.open("Mat.txt");
	chrono::time_point<std::chrono::system_clock> start, end;
	chrono::duration<double> elapsed_seconds;
	start = chrono::system_clock::now();
	
	if(argc == 4){
		inputA = argv[1];
		inputB = argv[2];
		outputC = argv[3];
	}else{
		printf("Incorrect number of arguments\n");
		cout << "Given: " << argc - 1 << " Expected: 3" << endl;
		return 0;
	}
	char buffer1[4];
	char buffer2[4];

    std::ifstream fileA;
    fileA.open(inputA,std::ios::binary);
	std::ifstream fileB;
	fileB.open(inputB,std::ios::binary);
    std::ofstream outC;
    outC.open(outputC);


    fileA.read(buffer1,4);
    int n1 = givint(buffer1);
    fileA.read(buffer1,4);
    int m1 = givint(buffer1);
    fileA.read(buffer1,4);
    int k1 = givint(buffer1);

	fileB.read(buffer2,4);
	int n2 = givint(buffer2);
	fileB.read(buffer2,4);
	int m2 = givint(buffer2);
	fileB.read(buffer2,4);
	int k2 = givint(buffer2);

	assert(n1 == n2);
	assert(m1 == m2);
	int n = n1, m = m1;

	vector<array<int,3>> keysa;
	vector<uint> vala;
	vector<array<int,3>> keysb;
	vector<uint> valb;

    // std::map<pair<int,int>,vector<uint>> inMapA;
    for (int i = 0; i < k1; ++i){
	    fileA.read(buffer1,4);
	    int a = givint(buffer1);
	    fileA.read(buffer1,4);
	    int b = givint(buffer1);
		array<int,3> pt = {a,b,i};
		keysa.push_back(pt);
	    vector<uint> vt;
	    for (int j = 0; j < m; ++j){
		  	for (int h = 0; h < m; ++h){
				int e = 0;
			  	fileA.read((char *)&e,2);
			    vt.push_back(e);
		  	}
	    }
		vala.insert(vala.end(),vt.begin(),vt.end());
    }
	fileA.close();
	// std::map<pair<int,int>,vector<uint>> inMapB;
	for (int i = 0; i < k2; ++i)
    {
	    fileB.read(buffer2,4);
	    int a = givint(buffer2);
	    fileB.read(buffer2,4);
	    int b = givint(buffer2);
		array<int,3> pt = {a,b,i};
		keysb.push_back(pt);
	    vector<uint> vt;
	    for (int j = 0; j < m; ++j){
		  	for (int h = 0; h < m; ++h){
				int e = 0;
			  	fileB.read((char *)&e,2);
			    vt.push_back(e);
		  	}
	    }
		valb.insert(valb.end(),vt.begin(),vt.end());
    }
	fileB.close();
    cout << "Input reading done, n: " << n << ", m: " << m <<  ", k-values: " << keysa.size() << " and " << keysb.size() << endl;
	
	sort(keysa.begin(), keysa.end());
	sort(keysb.begin(), keysb.end(), compmy);
	
	assert(keysa.size() == vala.size()/m/m);
	assert(keysb.size() == valb.size()/m/m);
	assert(keysa.size() == k1);
	assert(keysb.size() == k2);
	// inMapA.clear();
	// inMapB.clear();
	end = chrono::system_clock::now();
	elapsed_seconds = end-start;
	std::cout << "Sorted input: " << elapsed_seconds.count() << endl;

	// vector<pair<int,int>> keysc;
	vector<uint> valc((size_t)n*(size_t)n,0);
	start = chrono::system_clock::now();
	matMul(keysa, vala, keysb, valb, n, m, valc);
	end = chrono::system_clock::now();
	elapsed_seconds = end-start;
	std::cout << "Time taken for matrix multiplication: " << elapsed_seconds.count() << endl;

	// readable output
	int nm = n/m;
    cout << "Writing output" << endl;
    outC.write((char *)&n,4);
    outC.write((char *)&m,4);
    long kout = 0;
    outC.write((char *)&kout,4);
	for(int i = 0; i < nm; ++i)
	{	
		for (int j = 0; j < nm; ++j)
		{
			vector<uint> vtemp(valc.begin()+(i*nm + j)*m*m,valc.begin()+(i*nm + j + 1)*m*m);
			if(!isZero(&vtemp[0],m*m)){
				int aout = i;
				outC.write((char *)&aout,4);
				int bout = j;
				outC.write((char *)&bout,4);
				for (int k = 0; k < m*m; ++k){
					uint eout = vtemp[k];
					outC.write((char *)&eout,4);
				}
				kout += 1;
			}
		}
	}
    cout << "Number of output non-zero blocks: " << kout << endl;
    outC.seekp(8);
    outC.write((char *)&kout,4);
    cout << "Writing done" << endl;
    outC.close();
	// myFile.close();
	return 0;
}