#include "hip/hip_runtime.h"
#include "matrix.h"

using namespace std;
int main(int argc, char const *argv[])
{
	string inputA = "";
	string inputB = "";
	string outputC = "";

	// int thrds = 8;
	// std::ofstream myFile;
	// myFile.open("Mat.txt");
	chrono::time_point<std::chrono::system_clock> start, end;
	chrono::duration<double> elapsed_seconds;
	start = chrono::system_clock::now();
	
	if(argc == 4){
		inputA = argv[1];
		inputB = argv[2];
		outputC = argv[3];
	}else{
		printf("Incorrect number of arguments\n");
		cout << "Given: " << argc - 1 << " Expected: 3" << endl;
		return 0;
	}
	char buffer1[4];
	char buffer2[4];

    std::ifstream fileA;
    fileA.open(inputA,std::ios::binary);
	std::ifstream fileB;
	fileB.open(inputB,std::ios::binary);
    std::ofstream outC;
    outC.open(outputC);


    fileA.read(buffer1,4);
    int n1 = givint(buffer1);
    fileA.read(buffer1,4);
    int m1 = givint(buffer1);
    fileA.read(buffer1,4);
    int k1 = givint(buffer1);

	fileB.read(buffer2,4);
	int n2 = givint(buffer2);
	fileB.read(buffer2,4);
	int m2 = givint(buffer2);
	fileB.read(buffer2,4);
	int k2 = givint(buffer2);

	assert(n1 == n2);
	assert(m1 == m2);
	int n = n1, m = m1;

    std::map<pair<int,int>,vector<uint>> inMapA;
    for (long long i = 0; i < k1; ++i){
	    fileA.read(buffer1,4);
	    int a = givint(buffer1);
	    fileA.read(buffer1,4);
	    int b = givint(buffer1);
	    pair<int,int> pt = {a,b};
	    vector<uint> vt;
	    for (int j = 0; j < m; ++j){
		  	for (int h = 0; h < m; ++h){
				int e = 0;
			  	fileA.read((char *)&e,2);
			    vt.push_back(e);
		  	}
	    }
	    inMapA[pt] = vt; 
    }
	fileA.close();
	std::map<pair<int,int>,vector<uint>> inMapB;
	for (long long i = 0; i < k2; ++i)
    {
	    fileB.read(buffer2,4);
	    int a = givint(buffer2);
	    fileB.read(buffer2,4);
	    int b = givint(buffer2);
	    pair<int,int> pt = {a,b};
	    vector<uint> vt;
	    for (int j = 0; j < m; ++j){
		  	for (int h = 0; h < m; ++h){
				int e = 0;
			  	fileB.read((char *)&e,2);
			    vt.push_back(e);
		  	}
	    }
	    inMapB[pt] = vt; 
    }
	fileB.close();
    cout << "Input reading done, k values: " << inMapA.size() << " and " << inMapB.size() << endl;

	vector<pair<int,int>> keysa;
	vector<uint> vala;
	for(auto i = inMapA.begin(); i != inMapA.end(); i++){
		keysa.push_back(i->first);
		vala.insert(vala.end(),i->second.begin(),i->second.end());
	}

	vector<pair<int,int>> keysb;
	vector<uint> valb;
	for(auto i = inMapB.begin(); i != inMapB.end(); i++){
		keysb.push_back(i->first);
		valb.insert(valb.end(),i->second.begin(),i->second.end());
	}

	assert(keysa.size() == vala.size()/m/m);
	assert(keysb.size() == valb.size()/m/m);
	assert(keysa.size() == k1);
	assert(keysb.size() == k2);
	inMapA.clear();
	inMapB.clear();

	end = chrono::system_clock::now();
	elapsed_seconds = end-start;
	std::cout << "Map to vector conversion: " << elapsed_seconds.count() << endl;

	// vector<pair<int,int>> keysc;
	vector<uint> valc((size_t)n*(size_t)n,0);
	start = chrono::system_clock::now();
	matMul(keysa, vala, keysb, valb, n, m, valc);
	end = chrono::system_clock::now();
	elapsed_seconds = end-start;
	std::cout << "Time taken for matrix multiplication: " << elapsed_seconds.count() << endl;

	// readable output
	int nm = n/m;
    cout << "Writing output" << endl;
    outC.write((char *)&n,4);
    outC.write((char *)&m,4);
    long kout = 0;
    outC.write((char *)&kout,4);
	for(int i = 0; i < nm; ++i)
	{	
		for (int j = 0; j < nm; ++j)
		{
			vector<uint> vtemp(valc.begin()+(i*nm + j)*m*m,valc.begin()+(i*nm + j + 1)*m*m);
			if(!isZero(&vtemp[0],m*m)){
				int aout = i;
				outC.write((char *)&aout,4);
				int bout = j;
				outC.write((char *)&bout,4);
				for (int k = 0; k < m*m; ++k){
					uint eout = min(vtemp[k],(uint)MAX_VAL);
					outC.write((char *)&eout,4);
				}
				kout += 1;
			}
		}
	}

    cout << "Number of output non-zero blocks: " << kout << endl;
    outC.seekp(8);
    outC.write((char *)&kout,4);
    cout << "Writing done" << endl;
    outC.close();
	// myFile.close();
	return 0;
}