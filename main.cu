#include "hip/hip_runtime.h"
#include "matrix.h"

int main(int argc, char const *argv[])
{
	string inputA = "";
	string inputB = "";
	string outputC = "";

	// int thrds = 8;
	std::ofstream myFile;
	myFile.open("Mat.txt");
	chrono::time_point<std::chrono::system_clock> start, end;
	chrono::duration<double> elapsed_seconds;
	start = chrono::system_clock::now();
	
	if(argc == 4){
		inputA = argv[1];
		inputB = argv[2];
		outputC = argv[3];
	}else{
		printf("Incorrect number of arguments\n");
		cout << "Given: " << argc - 1 << " Expected: 3" << endl;
		return 0;
	}
	char buffer1[4];
	char buffer2[4];

    std::ifstream fileA;
    fileA.open(inputA,std::ios::binary);
	std::ifstream fileB;
	fileB.open(inputB,std::ios::binary);
    std::ofstream outC;
    outC.open(outputC);


    fileA.read(buffer1,4);
    int n1 = givint(buffer1);
    fileA.read(buffer1,4);
    int m1 = givint(buffer1);
    fileA.read(buffer1,4);
    int k1 = givint(buffer1);

	fileB.read(buffer2,4);
	int n2 = givint(buffer2);
	fileB.read(buffer2,4);
	int m2 = givint(buffer2);
	fileB.read(buffer2,4);
	int k2 = givint(buffer2);

	assert(n1 == n2);
	assert(m1 == m2);
	int n = n1, m = m1;

    std::map<pair<int,int>,vector<uint>> inMapA;
    for (int i = 0; i < k1; ++i){
	    fileA.read(buffer1,4);
	    int a = givint(buffer1);
	    fileA.read(buffer1,4);
	    int b = givint(buffer1);
	    pair<int,int> pt = {a,b};
	    vector<uint> vt;
	    for (int j = 0; j < m; ++j){
		  	for (int h = 0; h < m; ++h){
				int e = 0;
			  	fileA.read((char *)&e,2);
			    vt.push_back(e);
		  	}
	    }
	    inMapA[pt] = vt; 
    }
	fileA.close();
	std::map<pair<int,int>,vector<uint>> inMapB;
	for (int i = 0; i < k2; ++i)
    {
	    fileB.read(buffer2,4);
	    int a = givint(buffer2);
	    fileB.read(buffer2,4);
	    int b = givint(buffer2);
	    pair<int,int> pt = {a,b};
	    vector<uint> vt;
	    for (int j = 0; j < m; ++j){
		  	for (int h = 0; h < m; ++h){
				int e = 0;
			  	fileB.read((char *)&e,2);
			    vt.push_back(e);
		  	}
	    }
	    inMapB[pt] = vt; 
    }
	fileB.close();
    cout << "Input reading done, k values: " << inMapA.size() << " and " << inMapB.size() << endl;

	vector<pair<int,int>> keysa;
	for(auto i = inMapA.begin(); i != inMapA.end(); i++){
		keysa.push_back(i->first);
	}
	vector<uint> vala;
	for(auto i = inMapA.begin(); i != inMapA.end(); i++){
		vala.insert(vala.end(),i->second.begin(),i->second.end());
	}
	vector<pair<int,int>> keysb;
	for(auto i = inMapB.begin(); i != inMapB.end(); i++){
		keysb.push_back(i->first);
	}
	vector<uint> valb;
	for(auto i = inMapB.begin(); i != inMapB.end(); i++){
		valb.insert(valb.end(),i->second.begin(),i->second.end());
	}
	assert(keysa.size() == vala.size()/m/m);
	assert(keysb.size() == valb.size()/m/m);
	assert(keysa.size() == k1);
	assert(keysb.size() == k2);
	inMapA.clear();
	inMapB.clear();

	end = chrono::system_clock::now();
	elapsed_seconds = end-start;
	std::cout << "Map to vector conversion: " << elapsed_seconds.count() << endl;

	vector<pair<int,int>> keysc;
	vector<uint> valc;
	matMul(keysa, vala, keysb, valb, n, m, keysc, valc, myFile);
	end = chrono::system_clock::now();
	elapsed_seconds = end-start;
	std::cout << "Time taken for matrix multiplication: " << elapsed_seconds.count() << endl;


	std::map<pair<int,int>,vector<uint>> outMap;
	for (int i = 0; i < keysc.size(); ++i){
		outMap[keysc[i]] = vector<uint>(valc.begin()+i*m*m,valc.begin()+(i+1)*m*m);
	}
	keysc.clear();
	valc.clear();
	// readable output
	// printmap(inMapA,myFile,m);
	// printmap(inMapB,myFile,m);
    // printmap(outMap,myFile,m);

    cout << "Writing output" << endl;
    outC.write((char *)&n,4);
    outC.write((char *)&m,4);
    int kout = 0;
    outC.write((char *)&kout,4);
    for(auto i = outMap.begin(); i != outMap.end(); i++){
	    int aout = i->first.first;
	    outC.write((char *)&aout,4);
	    int bout = i->first.second;
	    outC.write((char *)&bout,4);
	    for (int j = 0; j < m; ++j){
		  	for (int h = 0; h < m; ++h){
			  	uint eout = min(i->second[j*m + h],(uint)MAX_VAL);
			  	outC.write((char *)&eout,4);
		  	}
	    }
	    kout += 1;
    }

    cout << "Number of output non-zero blocks: " << kout << endl;
    outC.seekp(8);
    outC.write((char *)&kout,4);
    cout << "Writing done" << endl;
    outC.close();
	myFile.close();
	return 0;
}