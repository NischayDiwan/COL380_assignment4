#include "hip/hip_runtime.h"
#include "matrix.h"

int main(int argc, char const *argv[])
{
	string inputA = "";
	string inputB = "";
	string outputC = "";

	// int thrds = 8;
	// std::ofstream myFile;
	// myFile.open("Mat.txt");
	chrono::time_point<std::chrono::system_clock> start, end;
	chrono::duration<double> elapsed_seconds;
	start = chrono::system_clock::now();
	
	if(argc == 4){
		inputA = argv[1];
		inputB = argv[2];
		outputC = argv[3];
	}else{
		printf("Incorrect number of arguments\n");
		cout << "Given: " << argc - 1 << " Expected: 3" << endl;
		return 0;
	}
	char buffer1[4];
	char buffer2[4];

    std::ifstream fileA;
    fileA.open(inputA,std::ios::binary);
	std::ifstream fileB;
	fileB.open(inputB,std::ios::binary);
    std::ofstream outC;
    outC.open(outputC);


    fileA.read(buffer1,4);
    int n1 = givint(buffer1);
    fileA.read(buffer1,4);
    int m1 = givint(buffer1);
    fileA.read(buffer1,4);
    int k1 = givint(buffer1);

	fileB.read(buffer2,4);
	int n2 = givint(buffer2);
	fileB.read(buffer2,4);
	int m2 = givint(buffer2);
	fileB.read(buffer2,4);
	int k2 = givint(buffer2);

	assert(n1 == n2);
	assert(m1 == m2);
	int n = n1, m = m1;

	vector<array<int,3>> keysa;
	vector<uint> vala;
	vector<array<int,3>> keysb;
	vector<uint> valb;

    // std::map<pair<int,int>,vector<uint>> inMapA;
    for (int i = 0; i < k1; ++i){
	    fileA.read(buffer1,4);
	    int a = givint(buffer1);
	    fileA.read(buffer1,4);
	    int b = givint(buffer1);
		array<int,3> pt = {a,b,i};
		keysa.push_back(pt);
	    vector<uint> vt;
	    for (int j = 0; j < m; ++j){
		  	for (int h = 0; h < m; ++h){
				int e = 0;
			  	fileA.read((char *)&e,2);
			    vt.push_back(e);
		  	}
	    }
		vala.insert(vala.end(),vt.begin(),vt.end());
    }
	fileA.close();
	// std::map<pair<int,int>,vector<uint>> inMapB;
	for (int i = 0; i < k2; ++i)
    {
	    fileB.read(buffer2,4);
	    int a = givint(buffer2);
	    fileB.read(buffer2,4);
	    int b = givint(buffer2);
		array<int,3> pt = {a,b,i};
		keysb.push_back(pt);
	    vector<uint> vt;
	    for (int j = 0; j < m; ++j){
		  	for (int h = 0; h < m; ++h){
				int e = 0;
			  	fileB.read((char *)&e,2);
			    vt.push_back(e);
		  	}
	    }
		valb.insert(valb.end(),vt.begin(),vt.end());
    }
	fileB.close();
    cout << "Input reading done, k values: " << keysa.size() << " and " << keysb.size() << endl;
	
	sort(keysa.begin(), keysa.end());
	sort(keysb.begin(), keysb.end());
	
	assert(keysa.size() == vala.size()/m/m);
	assert(keysb.size() == valb.size()/m/m);
	assert(keysa.size() == k1);
	assert(keysb.size() == k2);
	// inMapA.clear();
	// inMapB.clear();
	end = chrono::system_clock::now();
	elapsed_seconds = end-start;
	std::cout << "Sorted input: " << elapsed_seconds.count() << endl;

	// vector<pair<int,int>> keysc;
	vector<uint> valc((size_t)n*(size_t)n,0);
	start = chrono::system_clock::now();
	matMul(keysa, vala, keysb, valb, n, m, valc);
	end = chrono::system_clock::now();
	elapsed_seconds = end-start;
	std::cout << "Time taken for matrix multiplication: " << elapsed_seconds.count() << endl;

	// readable output
	int nm = n/m;
    cout << "Writing output" << endl;
    outC.write((char *)&n,4);
    outC.write((char *)&m,4);
    long kout = 0;
    outC.write((char *)&kout,4);
	for(int i = 0; i < nm; ++i)
	{	
		for (int j = 0; j < nm; ++j)
		{
			vector<uint> vtemp(valc.begin()+(i*nm + j)*m*m,valc.begin()+(i*nm + j + 1)*m*m);
			if(!isZero(&vtemp[0],m*m)){
				int aout = i;
				outC.write((char *)&aout,4);
				int bout = j;
				outC.write((char *)&bout,4);
				for (int k = 0; k < m*m; ++k){
					uint eout = min(vtemp[k],(uint)MAX_VAL);
					outC.write((char *)&eout,4);
				}
				kout += 1;
			}
		}
	}
    cout << "Number of output non-zero blocks: " << kout << endl;
    outC.seekp(8);
    outC.write((char *)&kout,4);
    cout << "Writing done" << endl;
    outC.close();
	// myFile.close();
	return 0;
}