#include "hip/hip_runtime.h"
#include "matrix.h"

void printvec(vector<uint> &a,ofstream &outstr, int m){
	for(int i = 0;i < a.size()/m;i++){
		for(int j=0; j < m; j++)
			outstr << a[i*m + j] << " ";
		outstr << endl;
	}
}

void printmap(map<pair<int,int>,vector<uint>> &mp, ofstream &outstr, int m){
	for(auto i = mp.begin(); i != mp.end(); i++){
		outstr << i->first.first << "," << i->first.second << ":\n";
		printvec((i->second),outstr,m);
	}
}

int givint(char *buffer){
	int *res = (int *)(buffer);
	return *res;
}

int binSearch(vector<pair<int,int>> &v, pair<int,int> p){
    auto it = lower_bound(v.begin(), v.end(), p);
	if(it != v.end()){
		int id = it - v.begin();
		if(v[id] == p)
			return id;
		else
			return -1;
	}
	return -1;
}

__device__
int binASearch(int *v, int l, int p1, int p2){
	// binary search in array
	int s = 0;
	int e = l;
	while(s <= e){
		int m = s + (e-s) / 2;
		if(v[2*m] == p1 && v[2*m+1] == p2)
			return m;
		if((v[2*m] < p1) || (v[2*m] == p1 && v[2*m+1] < p2))
			s = m + 1;
		else
			e = m - 1;
	}
	return -1;
}

bool isZero(uint *v, int l){
	bool flg = true;
	for (int i = 0; i < l; ++i)
	{	
		if(v[i] != 0){
			flg = false;
			break;
		}
	}
	return flg;
}

void inline transpose(vector<uint> &a,vector<uint> &b, int m){
	for(int i=0; i < m; i++){
		for(int j=0; j < m; j++){
			b[i*m + j] = a[j *m + i];
		}
	}
}

void inline blockOuter(vector<uint> &v1,vector<uint> &v2, vector<uint> &r, int m){
	for (int i = 0; i < v1.size(); ++i)
	{	
		r[i] = v1[i] + v2[i];
	}
}

void inline blockInner(vector<uint> &v1,vector<uint> &v2, vector<uint> &r, int m){
	for (int i = 0; i < m; ++i)
	{	
		for (int j = 0; j < m; ++j)
		{
			uint temp = 0;
			for (int k = 0; k < m; ++k)
			{
				temp = temp + (v1[i*m + k] * v2[k*m + j]);
			}
			r[i*m + j] = temp;
		}
	}
}

__global__
void matMulGPU(uint *a, uint *b, uint *c, int *ka, int *kb, int m, int n, int k1, int k2){
	extern __shared__ uint dab[];
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int nm = n/m;
	int i = bid / nm;
	int k = bid % nm;
	// int gtid = bid*blockDim.x + tid;
	uint temp = 0;
	for (int j = 0; j < nm; j++)
	{
		int id1 = binASearch(ka,k1,i,j);
		int id2 = binASearch(kb,k2,j,k);
		if(!(id1 == -1 || id2 == -1)){
			dab[tid] = a[tid + id1*m*m];
			dab[tid + m*m] = b[tid + id2*m*m];
			__syncthreads();
			
			int ii = tid/m;
			int jj = tid%m;
			for (int kk = 0; kk < m; ++kk)
			{
				temp = temp + (dab[ii*m + kk] * dab[kk*m + jj + m*m]);
			}
			__syncthreads();
		}
	}
	c[tid + i*m*n + k*m*m] = temp;
}

void matMul(vector<pair<int,int>> &mp1, vector<uint> &blksA, vector<pair<int,int>> &mp2, vector<uint> &blksB,  int n, int m, vector<pair<int,int>> &resm, vector<uint> &blksC, ofstream &outstr){
	int nm = n/m;

	// sending data to GPU
	int streamSize = 4;
	hipStream_t stream[streamSize];
	// hipError_t result[2];
	for(int i = 0;i<streamSize ;i++){
		hipStreamCreate(&stream[i]);
	}
	int size = sizeof(uint);
	uint *a = &blksA[0], *b = &blksB[0], *c = new uint[n*n]();
	uint *da, *db, *dc;
	hipMalloc(&da,size*blksA.size());
	hipMalloc(&db,size*blksB.size());
	hipMalloc(&dc,size*n*n);
	hipMemset(dc,0,size*n*n);
	int *ka, *kb;
	hipMalloc(&ka,2*sizeof(int)*mp1.size());
	hipMalloc(&kb,2*sizeof(int)*mp2.size());
	hipMemcpyAsync(da,a,size*blksA.size(),hipMemcpyHostToDevice,stream[0]);
	hipMemcpyAsync(db,b,size*blksB.size(),hipMemcpyHostToDevice,stream[1]);
	hipMemcpyAsync(ka,&mp1[0],2*sizeof(int)*mp1.size(),hipMemcpyHostToDevice,stream[2]);
	hipMemcpyAsync(kb,&mp2[0],2*sizeof(int)*mp2.size(),hipMemcpyHostToDevice,stream[3]);

	// converting to CSR
	vector<int> colV;
	vector<int> rofV;
	int offset = 0;
	int rowno = 0;
	rofV.push_back(offset);
	for(auto i = mp1.begin(); i != mp1.end(); i++){
		colV.push_back(i->second);
		if(i->first > rowno){
			for(int cc = 0;cc<(i->first-rowno);cc++){
				rofV.push_back(offset);
			}
			rowno = i->first;
		}
		offset+=1;
	}
	for(int j = rowno;j<nm;j++){
		rofV.push_back(blksA.size()/m/m);
	}
	// std::cout << "CSR converted\n";
	int *rof, *col;

	hipDeviceSynchronize();
	// std::cout << "gpu multiplication start\n";
	chrono::time_point<std::chrono::system_clock> startg = std::chrono::system_clock::now();
	// core matrix multiplication
	// for(int i = 0; i < nm; ++i)
	// {	
	// 	for (int k = 0; k < nm; ++k)
	// 	{
	// 		for (int j = rofV[i]; j < rofV[i+1]; j++)
	// 		{
	// 			int cl = colV[j];
	// 			int id = binASearch(&mp2[0].first,mp2.size(),{cl,k});
	// 			if(!(id == -1)){
	// 				matMulGPU<<<1,m*m,2*size*m*m>>>(j,id,da,db,dc,m,n,i,k);
	// 			}
	// 		}
	// 	}
	// }
	matMulGPU<<<nm*nm,m*m,2*size*m*m>>>(da,db,dc,ka,kb,m,n,mp1.size(),mp2.size()); // i X k

	hipDeviceSynchronize();
	// std::cout << "gpu multiplication done\n";
	chrono::time_point<std::chrono::system_clock> endg = std::chrono::system_clock::now();
	chrono::duration<double> elapsed_secondsg = endg-startg;
	std::cout << "gpu multiplication time: " << elapsed_secondsg.count() << "s\n";
	hipMemcpy(c,dc,size*n*n,hipMemcpyDeviceToHost);
	
	for(int i = 0; i < nm; ++i)
	{	
		for (int k = 0; k < nm; ++k)
		{
			vector<uint> vtemp(c+(i*nm + k)*m*m,c+(i*nm + k + 1)*m*m);
			if(!isZero(&vtemp[0],m*m)){
				pair<int,int> ptemp = {i,k};
				resm.push_back(ptemp);
				blksC.insert(blksC.end(),vtemp.begin(),vtemp.end());
			}
		}
	}

	// free the memory
	delete c;
	hipFree(da);
	hipFree(db);
	hipFree(dc);
	for(int i = 0;i<streamSize ;i++){
		hipStreamDestroy(stream[i]);
	}
}
