#include "hip/hip_runtime.h"
#include "matrix.h"

#define timer 0

void printvec(vector<uint> &a,ofstream &outstr, int m){
	for(int i = 0;i < a.size()/m;i++){
		for(int j=0; j < m; j++)
			outstr << a[i*m + j] << " ";
		outstr << endl;
	}
}

int givint(char *buffer){
	int *res = (int *)(buffer);
	return *res;
}

int binSearch(vector<pair<int,int>> &v, pair<int,int> p){
    auto it = lower_bound(v.begin(), v.end(), p);
	if(it != v.end()){
		int id = it - v.begin();
		if(v[id] == p)
			return id;
		else
			return -1;
	}
	return -1;
}

bool isZero(uint *v, int l){
	bool flg = true;
	for (int i = 0; i < l; ++i)
	{	
		if(v[i] != 0){
			flg = false;
			break;
		}
	}
	return flg;
}

__device__
int binASearch(int *v, int l, int p1, int p2){
	// binary search in array
	int s = 0;
	int e = l;
	while(s <= e){
		int m = s + (e-s) / 2;
		if(v[2*m] == p1 && v[2*m+1] == p2)
			return m;
		if((v[2*m] < p1) || (v[2*m] == p1 && v[2*m+1] < p2))
			s = m + 1;
		else
			e = m - 1;
	}
	return -1;
}

__global__
void matMulGPU(uint *a, uint *b, uint *c, int *ka, int *kb, int *rof, int *col, int m, int n, int k1, int k2){
	extern __shared__ uint dab[];
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int nm = n/m;
	int i = bid / nm;
	int k = bid % nm;
	// int gtid = bid*blockDim.x + tid;
	uint temp = 0;
	for (int j = rof[i]; j < rof[i+1]; j++){
		// int id1 = binASearch(ka,k1,i,j);
		int id1 = j;
		int id2 = binASearch(kb,k2,col[id1],k);
		if(!(id1 == -1 || id2 == -1)){
			dab[tid] = a[tid + id1*m*m];
			dab[tid + m*m] = b[tid + id2*m*m];
			__syncthreads();
			
			int ii = tid/m;
			int jj = tid%m;
			for (int kk = 0; kk < m; ++kk)
			{
				temp = temp + (dab[ii*m + kk] * dab[kk*m + jj + m*m]);
			}
			__syncthreads();
		}
	}
	c[tid + i*m*n + k*m*m] = temp;
}

void matMul(vector<pair<int,int>> &mp1, vector<uint> &blksA, vector<pair<int,int>> &mp2, vector<uint> &blksB,  int n, int m, vector<uint> &blksC){
	int nm = n/m;

	// sending data to GPU
	int streamSize = 6;
	hipStream_t stream[streamSize];
	for(int i = 0;i<streamSize ;i++){
		hipStreamCreate(&stream[i]);
	}
	int size = sizeof(uint);
	uint *a = &blksA[0], *b = &blksB[0], *c = &blksC[0];
	uint *da, *db, *dc;
	hipMalloc(&da,size*blksA.size());
	hipMalloc(&db,size*blksB.size());
	hipMalloc(&dc,size*n*n);
	hipMemset(dc,0,size*n*n);
	int *ka, *kb;
	hipMalloc(&ka,2*sizeof(int)*mp1.size());
	hipMalloc(&kb,2*sizeof(int)*mp2.size());
	hipMemcpyAsync(da,a,size*blksA.size(),hipMemcpyHostToDevice,stream[0]);
	hipMemcpyAsync(db,b,size*blksB.size(),hipMemcpyHostToDevice,stream[1]);
	hipMemcpyAsync(ka,&mp1[0],2*sizeof(int)*mp1.size(),hipMemcpyHostToDevice,stream[2]);
	hipMemcpyAsync(kb,&mp2[0],2*sizeof(int)*mp2.size(),hipMemcpyHostToDevice,stream[3]);

	// converting to CSR
	vector<int> colV;
	vector<int> rofV;
	int offset = 0;
	int rowno = 0;
	rofV.push_back(offset);
	for(auto i = mp1.begin(); i != mp1.end(); i++){
		colV.push_back(i->second);
		if(i->first > rowno){
			for(int cc = 0;cc<(i->first-rowno);cc++){
				rofV.push_back(offset);
			}
			rowno = i->first;
		}
		offset+=1;
	}
	for(int j = rowno;j<nm;j++){
		rofV.push_back(blksA.size()/m/m);
	}
	// std::cout << "CSR converted\n";
	int *rof, *col;
	hipMalloc(&rof,rofV.size()*sizeof(int));
	hipMalloc(&col,colV.size()*sizeof(int));
	hipMemcpyAsync(rof,&rofV[0],rofV.size()*sizeof(int),hipMemcpyHostToDevice,stream[4]);
	hipMemcpyAsync(col,&colV[0],colV.size()*sizeof(int),hipMemcpyHostToDevice,stream[5]);
	
	if(timer){
		hipDeviceSynchronize();
		chrono::time_point<std::chrono::system_clock> startg = std::chrono::system_clock::now();
		matMulGPU<<<nm*nm,m*m,2*size*m*m>>>(da,db,dc,ka,kb,rof,col,m,n,mp1.size(),mp2.size()); // i X k
		hipDeviceSynchronize();
		chrono::time_point<std::chrono::system_clock> endg = std::chrono::system_clock::now();
		chrono::duration<double> elapsed_secondsg = endg-startg;
		std::cout << "gpu multiplication time: " << elapsed_secondsg.count() << "s\n";
	}else{
		hipDeviceSynchronize();
		matMulGPU<<<nm*nm,m*m,2*size*m*m>>>(da,db,dc,ka,kb,rof,col,m,n,mp1.size(),mp2.size()); // i X k
	}

	int chunk = n*n/4;
	hipMemcpyAsync(c,dc,chunk*size,hipMemcpyDeviceToHost,stream[0]);
	hipMemcpyAsync(c+chunk,dc+chunk,size*chunk,hipMemcpyDeviceToHost,stream[1]);
	hipMemcpyAsync(c+2*chunk,dc+2*chunk,size*chunk,hipMemcpyDeviceToHost,stream[2]);
	hipMemcpyAsync(c+3*chunk,dc+3*chunk,size*chunk,hipMemcpyDeviceToHost,stream[3]);
	hipDeviceSynchronize();

	// free the memory
	hipFree(da);
	hipFree(db);
	hipFree(ka);
	hipFree(kb);
	hipFree(rof);
	hipFree(col);
	hipFree(dc);
	for(int i = 0;i<streamSize ;i++){
		hipStreamDestroy(stream[i]);
	}
}
