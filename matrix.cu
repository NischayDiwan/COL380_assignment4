#include "hip/hip_runtime.h"
#include "matrix.h"

using namespace std;
#define timer 0

void printvec(vector<uint> &a,ofstream &outstr, int m){
	for(int i = 0;i < a.size()/m;i++){
		for(int j=0; j < m; j++)
			outstr << a[i*m + j] << " ";
		outstr << endl;
	}
}

int givint(char *buffer){
	int *res = (int *)(buffer);
	return *res;
}

int binSearch(vector<pair<int,int>> &v, pair<int,int> p){
    auto it = lower_bound(v.begin(), v.end(), p);
	if(it != v.end()){
		int id = it - v.begin();
		if(v[id] == p)
			return id;
		else
			return -1;
	}
	return -1;
}

bool isZero(uint *v, int l){
	bool flg = true;
	for (int i = 0; i < l; ++i)
	{	
		if(v[i] != 0){
			flg = false;
			break;
		}
	}
	return flg;
}

__device__
int binASearch(int *v, int l, int p1, int p2){
	// binary search in array
	int s = 0;
	int e = l;
	while(s <= e){
		int m = s + (e-s) / 2;
		if(v[2*m] == p1 && v[2*m+1] == p2)
			return m;
		if((v[2*m] < p1) || (v[2*m] == p1 && v[2*m+1] < p2))
			s = m + 1;
		else
			e = m - 1;
	}
	return -1;
}

__global__
void matMulGPU(uint *a, uint *b, uint *c, int *ka, int *kb, int *rof, int *col, int m, int n, int k1, int k2){
	extern __shared__ uint dab[];
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int nm = n/m;
	int i = bid / nm;
	int k = bid % nm;
	uint temp = 0;
	for (int j = rof[i]; j < rof[i+1]; j++){
		// int id1 = binASearch(ka,k1,i,j);
		int id1 = j;
		int id2 = binASearch(kb,k2,col[id1],k);
		if(!(id1 == -1 || id2 == -1)){
			dab[tid] = a[tid + id1*m*m];
			dab[tid + m*m] = b[tid + id2*m*m];
			__syncthreads();
			
			int ii = tid/m;
			int jj = tid%m;
			for (int kk = 0; kk < m; ++kk)
			{
				temp = temp + (dab[ii*m + kk] * dab[kk*m + jj + m*m]);
			}
			__syncthreads();
		}
		__syncthreads();
	}
	__syncthreads();
	c[tid + i*m*n + k*m*m] = temp;

	// if(temp != 0)
	// 	printf("%d\n",c[tid + i*m*n + k*m*m]);
}

void matMul(vector<pair<int,int>> &mp1, vector<uint> &blksA, vector<pair<int,int>> &mp2, vector<uint> &blksB,  long long n, long long m, vector<uint> &blksC){
	long long nm = n/m;

	// sending data to GPU
	int streamSize = 6;
	hipError_t err;
	hipStream_t stream[streamSize];
	for(int i = 0;i<streamSize ;i++){
		hipStreamCreate(&stream[i]);
	}
	size_t size = sizeof(uint);
	size_t size2 = sizeof(int);
	size_t size3 = size * n * n;
	uint *a = blksA.data(), *b = blksB.data(), *c = blksC.data();
	uint *da, *db, *dc;
	hipMalloc(&da,size*blksA.size());
	hipMalloc(&db,size*blksB.size());
	hipMalloc(&dc,size3);
	hipDeviceSynchronize();
	hipMemset(dc,0,size3);
	int *ka, *kb;
	hipMalloc(&ka,(size_t)2*size2*(size_t)mp1.size());
	hipMalloc(&kb,(size_t)2*size2*(size_t)mp2.size());
	hipMemcpyAsync(da,a,(size_t)size*(size_t)blksA.size(),hipMemcpyHostToDevice,stream[0]);
	hipMemcpyAsync(db,b,(size_t)size*(size_t)blksB.size(),hipMemcpyHostToDevice,stream[1]);
	hipMemcpyAsync(ka,&mp1[0],(size_t)2*size2*(size_t)mp1.size(),hipMemcpyHostToDevice,stream[2]);
	hipMemcpyAsync(kb,&mp2[0],(size_t)2*size2*(size_t)mp2.size(),hipMemcpyHostToDevice,stream[3]);

	// converting to CSR
	vector<int> colV;
	vector<int> rofV;
	int offset = 0;
	int rowno = 0;
	rofV.push_back(offset);
	for(auto i = mp1.begin(); i != mp1.end(); i++){
		colV.push_back(i->second);
		if(i->first > rowno){
			for(int cc = 0;cc<(i->first-rowno);cc++){
				rofV.push_back(offset);
			}
			rowno = i->first;
		}
		offset+=1;
	}
	for(int j = rowno;j<nm;j++){
		rofV.push_back(blksA.size()/m/m);
	}
	// std::cout << "CSR converted\n";
	int *rof, *col;
	hipMalloc(&rof,(size_t)rofV.size()*size2);
	hipMalloc(&col,(size_t)colV.size()*size2);
	hipMemcpyAsync(rof,&rofV[0],(size_t)rofV.size()*size2,hipMemcpyHostToDevice,stream[4]);
	hipMemcpyAsync(col,&colV[0],(size_t)colV.size()*size2,hipMemcpyHostToDevice,stream[5]);

	int stride = (int)(nm*nm);
	if(timer){
		hipDeviceSynchronize();
		chrono::time_point<std::chrono::system_clock> startg = std::chrono::system_clock::now();
		matMulGPU<<<stride,m*m,2*size*m*m,0>>>(da,db,dc,ka,kb,rof,col,m,n,mp1.size(),mp2.size()); // i X k
		hipDeviceSynchronize();
		chrono::time_point<std::chrono::system_clock> endg = std::chrono::system_clock::now();
		chrono::duration<double> elapsed_secondsg = endg-startg;
		std::cout << "gpu multiplication time: " << elapsed_secondsg.count() << "s\n";
	}else{
		hipDeviceSynchronize();
		matMulGPU<<<stride,m*m,2*size*m*m,0>>>(da,db,dc,ka,kb,rof,col,m,n,mp1.size(),mp2.size()); // i X k
	}

	hipDeviceSynchronize();
	err = hipMemcpy((void *)c,(void *)dc,size3,hipMemcpyDeviceToHost);
	if (err != hipSuccess) 
	    printf("Error: %s\n", hipGetErrorString(err));

	// free the memory
	hipFree(da);
	hipFree(db);
	hipFree(ka);
	hipFree(kb);
	hipFree(rof);
	hipFree(col);
	hipFree(dc);
	for(int i = 0;i<streamSize ;i++){
		hipStreamDestroy(stream[i]);
	}
}
