#include "hip/hip_runtime.h"
#include "matrix.h"

void printvec(vector<uint> &a,ofstream &outstr, int m){
	for(int i = 0;i < a.size()/m;i++){
		for(int j=0; j < m; j++)
			outstr << a[i*m + j] << " ";
		outstr << endl;
	}
}

void printmap(map<pair<int,int>,vector<uint>> &mp, ofstream &outstr, int m){
	for(auto i = mp.begin(); i != mp.end(); i++){
		outstr << i->first.first << "," << i->first.second << ":\n";
		printvec((i->second),outstr,m);
	}
}

int givint(char *buffer){
	int *res = (int *)(buffer);
	return *res;
}

int binSearch(vector<pair<int,int>> &v, pair<int,int> p){
    auto it = lower_bound(v.begin(), v.end(), p);
	if(it != v.end()){
		int id = it - v.begin();
		if(v[id] == p)
			return id;
		else
			return -1;
	}
	return -1;
}

void inline transpose(vector<uint> &a,vector<uint> &b, int m){
	for(int i=0; i < m; i++){
		for(int j=0; j < m; j++){
			b[i*m + j] = a[j *m + i];
		}
	}
}

bool inline isZero(vector<uint> &v){
	bool flg = true;
	for (int i = 0; i < v.size(); ++i)
	{	
		if(v[i] != 0){
			flg = false;
			break;
		}
	}
	return flg;
}

void inline blockOuter(vector<uint> &v1,vector<uint> &v2, vector<uint> &r, int m){
	for (int i = 0; i < v1.size(); ++i)
	{	
		r[i] = v1[i] + v2[i];
	}
}

void inline blockInner(vector<uint> &v1,vector<uint> &v2, vector<uint> &r, int m){
	for (int i = 0; i < m; ++i)
	{	
		for (int j = 0; j < m; ++j)
		{
			uint temp = 0;
			for (int k = 0; k < m; ++k)
			{
				temp = temp + (v1[i*m + k] * v2[k*m + j]);
			}
			r[i*m + j] = temp;
		}
	}
}

__global__
void matMulGPU(uint *a, uint *b, uint *c, int m){
	extern __shared__ uint dab[];
	// int bid = blockIdx.x;
	int tid = threadIdx.x;
	// int gtid = bid*blockDim.x + tid;
	dab[tid] = a[tid];
	dab[tid + m*m] = b[tid];
	__syncthreads();
	uint temp = 0;
	int i = tid/m;
	int j = tid - i*m;
	for (int k = 0; k < m; ++k)
	{
		temp = temp + (dab[i*m + k] * dab[k*m + j + m*m]);
	}
	// __syncthreads();
	c[tid] = temp;
}

void inline blockGPUmul(vector<uint> &v1,vector<uint> &v2, vector<uint> &r, int m){
	uint *a = &v1[0], *b = &v2[0], *c = &r[0];
	uint *da, *db, *dc;
	int size = m*m*sizeof(int);
	hipMalloc(&da,size);
	hipMalloc(&db,size);
	hipMalloc(&dc,size);
	hipMemcpy(da,a,size,hipMemcpyHostToDevice);
	hipMemcpy(db,b,size,hipMemcpyHostToDevice);
	matMulGPU<<<1,m*m,2*size>>>(da,db,dc,m);
	// hipDeviceSynchronize();
	hipMemcpy(c,dc,size,hipMemcpyDeviceToHost);
	hipFree(da);
	hipFree(db);
	hipFree(dc);
}

void matMul(vector<pair<int,int>> &mp1, vector<uint> &blksA, vector<pair<int,int>> &mp2, vector<uint> &blksB,  int n, int m, vector<pair<int,int>> &resm, vector<uint> &blksC, ofstream &outstr){
	int nm = n/m;
	
	vector<uint> valV(blksA.begin(),blksA.end());
	vector<uint> colV;
	vector<uint> rofV;
	// vector<uint> rowV;
	int offset = 0;
	int rowno = 0;
	rofV.push_back(offset);
	for(auto i = mp1.begin(); i != mp1.end(); i++){
		colV.push_back(i->second);
		if(i->first > rowno){
			for(int cc = 0;cc<(i->first-rowno);cc++){
				rofV.push_back(offset);
			}
			rowno = i->first;
		}
		offset+=1;
	}
	for(int j = rowno;j<nm;j++){
		rofV.push_back(valV.size()/m/m);
	}

	for(int i = 0; i < nm; ++i)
	{	
		for (int k = 0; k < nm; ++k)
		{
			vector<uint> vtemp(m*m,0);
			for (int j = rofV[i]; j < rofV[i+1]; j++)
			{
				int cl = colV[j];
				if(!(binSearch(mp2,{cl,k}) == -1)){
					vector<uint> vtemp1(m*m,0);
					vector<uint> tval(valV.begin() + j*m*m, valV.begin() + (j+1)*m*m);
					int id = binSearch(mp2,{cl,k});
					vector<uint> tval1(blksB.begin() + id*m*m, blksB.begin() + (id+1)*m*m);
					blockInner(tval,tval1,vtemp1,m);
					// blockGPUmul(valV[j],mp2[{cl,k}],vtemp1,m);
					blockOuter(vtemp,vtemp1,vtemp,m);
				}
			}
			if(!isZero(vtemp)){
				pair<int,int> ptemp = {i,k};
				resm.push_back(ptemp);
				blksC.insert(blksC.end(),vtemp.begin(),vtemp.end());
			}
		}
	}
}
