#include "hip/hip_runtime.h"
#include "matrix.h"

void printvec(vector<uint> &a,ofstream &outstr, int m){
	for(int i = 0;i < a.size()/m;i++){
		for(int j=0; j < m; j++)
			outstr << a[i*m + j] << " ";
		outstr << endl;
	}
}

void printmap(map<pair<int,int>,vector<uint>> &mp, ofstream &outstr, int m){
	for(auto i = mp.begin(); i != mp.end(); i++){
		outstr << i->first.first << "," << i->first.second << ":\n";
		printvec((i->second),outstr,m);
	}
}

int givint(char *buffer){
	int *res = (int *)(buffer);
	return *res;
}

int binSearch(vector<pair<int,int>> &v, pair<int,int> p){
    auto it = lower_bound(v.begin(), v.end(), p);
	if(it != v.end()){
		int id = it - v.begin();
		if(v[id] == p)
			return id;
		else
			return -1;
	}
	return -1;
}

int binASearch(int *v, int l, pair<int,int> p){
	// binary search in array
	return -1;
}

bool isZero(uint *v, int l){
	bool flg = true;
	for (int i = 0; i < l; ++i)
	{	
		if(v[i] != 0){
			flg = false;
			break;
		}
	}
	return flg;
}

void inline transpose(vector<uint> &a,vector<uint> &b, int m){
	for(int i=0; i < m; i++){
		for(int j=0; j < m; j++){
			b[i*m + j] = a[j *m + i];
		}
	}
}

void inline blockOuter(vector<uint> &v1,vector<uint> &v2, vector<uint> &r, int m){
	for (int i = 0; i < v1.size(); ++i)
	{	
		r[i] = v1[i] + v2[i];
	}
}

void inline blockInner(vector<uint> &v1,vector<uint> &v2, vector<uint> &r, int m){
	for (int i = 0; i < m; ++i)
	{	
		for (int j = 0; j < m; ++j)
		{
			uint temp = 0;
			for (int k = 0; k < m; ++k)
			{
				temp = temp + (v1[i*m + k] * v2[k*m + j]);
			}
			r[i*m + j] = temp;
		}
	}
}

__global__
void matMulGPU(int id1, int id2, uint *a, uint *b, uint *c, int m, int rx1, int rx2){
	extern __shared__ uint dab[];
	// int bid = blockIdx.x;
	int tid = threadIdx.x;
	// int gtid = bid*blockDim.x + tid;
	dab[tid] = a[tid + id1*m*m];
	dab[tid + m*m] = b[tid + id2*m*m];
	__syncthreads();
	uint temp = 0;
	int i = tid/m;
	int j = tid - i*m;
	for (int k = 0; k < m; ++k)
	{
		temp = temp + (dab[i*m + k] * dab[k*m + j + m*m]);
	}
	// __syncthreads();
	c[tid] = temp;
}

void matMul(vector<pair<int,int>> &mp1, vector<uint> &blksA, vector<pair<int,int>> &mp2, vector<uint> &blksB,  int n, int m, vector<pair<int,int>> &resm, vector<uint> &blksC, ofstream &outstr){
	int nm = n/m;

	// sending data to GPU
	hipStream_t stream[2];
	// hipError_t result[2];
	for(int i = 0;i<2;i++){
		hipStreamCreate(&stream[i]);
	}
	int size = m*m*sizeof(int);
	uint *a = &blksA[0], *b = &blksB[0], *c = &blksC[0];
	uint *da, *db, *dc;
	hipMalloc(&da,size*blksA.size()/m/m);
	hipMalloc(&db,size*blksB.size()/m/m);
	hipMalloc(&dc,size);
	hipMemcpyAsync(da,a,size*blksA.size()/m/m,hipMemcpyHostToDevice,stream[0]);
	hipMemcpyAsync(db,b,size*blksB.size()/m/m,hipMemcpyHostToDevice,stream[1]);

	// converting to CSR
	vector<int> colV;
	vector<int> rofV;
	int offset = 0;
	int rowno = 0;
	rofV.push_back(offset);
	for(auto i = mp1.begin(); i != mp1.end(); i++){
		colV.push_back(i->second);
		if(i->first > rowno){
			for(int cc = 0;cc<(i->first-rowno);cc++){
				rofV.push_back(offset);
			}
			rowno = i->first;
		}
		offset+=1;
	}
	for(int j = rowno;j<nm;j++){
		rofV.push_back(blksA.size()/m/m);
	}
	cout << "CSR converted\n";

	// thrust::device_vector<uint> dblksA(blksA.begin(), blksA.end());
	// thrust::device_vector<uint> dblksB(blksB.begin(), blksB.end());
	// thrust::device_vector<uint> dblksC(n*n,0);
	// thrust::copy(blksA.begin(), blksA.end(), dblksA.begin());
	// thrust::copy(blksB.begin(), blksB.end(), dblksB.begin());

	hipDeviceSynchronize();
	cout << "copy done\n";
	// core matrix multiplication
	for(int i = 0; i < nm; ++i)
	{	
		for (int k = 0; k < nm; ++k)
		{
			vector<uint> vtemp(m*m,0);
			for (int j = rofV[i]; j < rofV[i+1]; j++)
			{
				int cl = colV[j];
				if(!(binSearch(mp2,{cl,k}) == -1)){
					int id = binSearch(mp2,{cl,k});
					vector<uint> vtemp1(m*m,0);
					c = &vtemp1[0];
					matMulGPU<<<1,m*m,2*size>>>(j,id,da,db,dc,m,i,k);
					// hipDeviceSynchronize();
					hipMemcpy(c,dc,size,hipMemcpyDeviceToHost);
					// blockInner(tval,tval1,vtemp1,m);
					blockOuter(vtemp,vtemp1,vtemp,m);
				}
			}
			if(!isZero(&vtemp[0],m*m)){
				pair<int,int> ptemp = {i,k};
				resm.push_back(ptemp);
				blksC.insert(blksC.end(),vtemp.begin(),vtemp.end());
			}
		}
	}
	// thrust::host_vector<uint> hblksC(dblksC.size());
	// thrust::copy(dblksC.begin(), dblksC.end(), hblksC.begin());
	// for(int i = 0; i < nm; ++i)
	// {	
	// 	for (int k = 0; k < nm; ++k)
	// 	{
	// 		vector<uint> vtemp(hblksC.begin() + (i*nm + k)*m*m, hblksC.begin() + (i*nm + k + 1)*m*m);
	// 		if(!isZero(vtemp)){
	// 			pair<int,int> ptemp = {i,k};
	// 			resm.push_back(ptemp);
	// 			blksC.insert(blksC.end(),vtemp.begin(),vtemp.end());
	// 		}
	// 	}
	// }

	// free the cuda memory
	hipFree(da);
	hipFree(db);
	hipFree(dc);
	for(int i = 0;i<2;i++){
		hipStreamDestroy(stream[i]);
	}
}
