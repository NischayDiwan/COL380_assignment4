#include "hip/hip_runtime.h"
#include "matrix.h"

void printvec(vector<uint> &a,ofstream &outstr){
   	for(int i=0; i < a.size(); i++)
		outstr << a.at(i) << " ";
	outstr << endl;
}

void printmap(map<pair<int,int>,vector<uint>> &m, ofstream &outstr){
	for(auto i = m.begin(); i != m.end(); i++){
		outstr << i->first.first << "," << i->first.second << ":\n";
		printvec((i->second),outstr);
	}
}

void inline transpose(vector<uint> &a,vector<uint> &b, int m){
	for(int i=0; i < m; i++){
		for(int j=0; j < m; j++){
			b[i*m + j] = a[j *m + i];
		}
	}
}

int givint(char *buffer){
	int *res = (int *)(buffer);
	return *res;
}

void inline blockOuter(vector<uint> &v1,vector<uint> &v2, vector<uint> &r, int m){
	for (int i = 0; i < v1.size(); ++i)
	{	
		r[i] = v1[i] + v2[i];
	}
}

void inline blockInner(vector<uint> &v1,vector<uint> &v2, vector<uint> &r, int m){
	for (int i = 0; i < m; ++i)
	{	
		for (int j = 0; j < m; ++j)
		{
			uint temp = 0;
			for (int k = 0; k < m; ++k)
			{
				temp = temp + (v1[i*m + k] * v2[k*m + j]);
			}
			r[i*m + j] = temp;
		}
	}
}

bool inline isZero(vector<uint> &v){
	bool flg = true;
	for (int i = 0; i < v.size(); ++i)
	{	
		if(v[i] != 0){
			flg = false;
			break;
		}
	}
	return flg;
}

__global__
void matMulGPU(void){
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int gtid = bid*blockDim.x + tid;
}

void matMul(map<pair<int,int>,vector<uint>> &mp1, map<pair<int,int>,vector<uint>> &mp2, int n, int m, int k1, int k2, map<pair<int,int>,vector<uint>> &resm){
	int nm = n/m;
	matMulGPU<<<m,m>>>();

	vector<vector<uint>> valV;
	vector<uint> colV;
	vector<uint> rofV;
	// vector<uint> rowV;
	int offset = 0;
	int rowno = 0;
	rofV.push_back(offset);
	for(auto i = mp1.begin(); i != mp1.end(); i++){
		valV.push_back(i->second);
		colV.push_back(i->first.second);
		// rowV.push_back(i->first.first);
		if(i->first.first > rowno){
			for(int cc = 0;cc<(i->first.first-rowno);cc++){
				rofV.push_back(offset);
			}
			rowno = i->first.first;
		}
		offset+=1;
	}
	for(int j = rowno;j<nm;j++){
		rofV.push_back(valV.size());
	}	
	for(int i = 0; i < nm; ++i)
	{	
		for (int k = 0; k < nm; ++k)
		{
			vector<uint> vtemp(m*m,0);
			for (int j = rofV[i]; j < rofV[i+1]; j++)
			{
				int cl = colV[j];
				if(!(mp2.find({cl,k}) == mp2.end())){
					vector<uint> vtemp1(m*m,0);
					blockInner(valV[j],mp2[{cl,k}],vtemp1,m);
					blockOuter(vtemp,vtemp1,vtemp,m);
				}
			}
			if(!isZero(vtemp)){
				pair<int,int> ptemp = {i,k};
				resm[ptemp] = vtemp;
			}
		}
	}
}
