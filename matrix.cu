#include <hip/hip_runtime.h>

#include <bits/stdc++.h>
#include <chrono>

#define MAX_VAL 4294967295
#define BLOCK_SIZE1 16
#define BLOCK_SIZE2 64
#define BLOCK_SIZEGPU blockDim.x
#define timer 0

using namespace std;
void printvec(vector<uint> &a,ofstream &outstr, int m){
	for(int i = 0;i < a.size()/m;i++){
		for(int j=0; j < m; j++)
			outstr << a[i*m + j] << " ";
		outstr << endl;
	}
}

int givint(char *buffer){
	int *res = (int *)(buffer);
	return *res;
}

bool isZero(uint *v, int l){
	bool flg = true;
	for (int i = 0; i < l; ++i)
	{	
		if(v[i] != 0){
			flg = false;
			break;
		}
	}
	return flg;
}

__device__
int binASearch(int *v, int l, int p1, int p2){
	// binary search in array
	int s = 0;
	int e = l;
	while(s <= e){
		int m = s + (e-s) / 2;
		if(v[3*m] == p1 && v[3*m+1] == p2)
			return v[3*m +2];
		if((v[3*m] < p1) || (v[3*m] == p1 && v[3*m+1] < p2))
			s = m + 1;
		else
			e = m - 1;
	}
	return -1;
}

__global__
void matMulGPU(uint *a, uint *b, uint *c, int *ka, int *kb, int *rof, int *col, int m, int n, int k1, int k2){
	extern __shared__ uint dab[];
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int nm = n/m;
	int i = bid / nm;
	int k = bid % nm;
	uint64_t temp = 0;
	// if(tid == 0 && bid == 0){
	// for (int it = 0; it < k1; it++)
	// {
	// 	printf("%d %d %d\n",ka[3*it],ka[3*it+1],ka[3*it+2]);
	// }
	// printf("-------------------\n");
	// for (int it = 0; it < k2; it++)
	// {
	// 	printf("%d %d %d\n",kb[3*it],kb[3*it+1],kb[3*it+2]);
	// }
	// printf("-------------------\n");
	// }
	for (int j = rof[i]; j < rof[i+1]; j++){
		// int id1 = binASearch(ka,k1,i,j);
		int id1 = ka[j];
		int cl = col[j];
		int id2 = binASearch(kb,k2,cl,k);
		// if(tid == 0 && bid == 0)
		// 	printf("%d %d : %d %d : %d %d\n",i,cl,cl,k,id1,id2);
		if(!(id2 == -1)){
			dab[tid] = (uint)a[tid + id1*m*m];
			dab[tid + m*m] = (uint)b[tid + id2*m*m];
			__syncthreads();
			
			int ii = tid/m;
			int jj = tid%m;
			for (int kk = 0; kk < m; ++kk)
			{
				temp = temp + (uint64_t)(dab[ii*m + kk] * dab[kk*m + jj + m*m]);
			}
			__syncthreads();
		}
		__syncthreads();
	}
	__syncthreads();
	c[tid + i*m*n + k*m*m] = min(temp,MAX_VAL);
}

void matMul(vector<array<int,3>> &mp1, vector<uint> &blksA, vector<array<int,3>> &mp2, vector<uint> &blksB,  long long n, long long m, vector<uint> &blksC){
	long long nm = n/m;
	// sending data to GPU
	int streamSize = 6;
	hipError_t err;
	hipStream_t stream[streamSize];
	for(int i = 0;i<streamSize ;i++){
		hipStreamCreate(&stream[i]);
	}
	size_t size = sizeof(uint);
	size_t size2 = sizeof(int);
	size_t size3 = sizeof(uint) * n * n;
	uint *a = blksA.data(), *b = blksB.data(), *da, *db;
	uint *c = blksC.data(), *dc;
	hipMalloc(&da,size*blksA.size());
	hipMalloc(&db,size*blksB.size());
	hipMalloc(&dc,size3);
	hipDeviceSynchronize();
	hipMemset(dc,0,size3);
	hipMemcpyAsync(da,a,(size_t)size*(size_t)blksA.size(),hipMemcpyHostToDevice,stream[0]);
	hipMemcpyAsync(db,b,(size_t)size*(size_t)blksB.size(),hipMemcpyHostToDevice,stream[1]);
	// int *ka;
	// cudaMalloc(&ka,(size_t)3*size2*(size_t)mp1.size());
	// cudaMemcpyAsync(ka,mp1.data(),(size_t)3*size2*(size_t)mp1.size(),cudaMemcpyHostToDevice,stream[2]);
	int *kb;
	hipMalloc(&kb,(size_t)3*size2*(size_t)mp2.size());
	hipMemcpyAsync(kb,mp2.data(),(size_t)3*size2*(size_t)mp2.size(),hipMemcpyHostToDevice,stream[3]);
	// err = cudaGetLastError();
	// if (err != cudaSuccess) 
	//     printf("Error: %s\n", cudaGetErrorString(err));

	// converting to CSR
	vector<int> valV;
	vector<int> colV;
	vector<int> rofV;
	int offset = 0;
	int rowno = 0;
	rofV.push_back(offset);
	for(int i = 0; i <mp1.size() ; i++){
		colV.push_back(mp1[i][1]);
		valV.push_back(mp1[i][2]);
		if(mp1[i][0] > rowno){
			for(int cc = 0;cc<(mp1[i][0]-rowno);cc++){
				rofV.push_back(offset);
			}
			rowno = mp1[i][0];
		}
		offset+=1;
	}
	for(int j = rowno;j<nm;j++){
		rofV.push_back(blksA.size()/m/m);
	}

	// std::cout << "CSR converted\n";
	int *ka;
	hipMalloc(&ka,(size_t)size2*(size_t)mp1.size());
	hipMemcpyAsync(ka,valV.data(),(size_t)size2*(size_t)mp1.size(),hipMemcpyHostToDevice,stream[2]);
	int *rof, *col;
	hipMalloc(&rof,(size_t)rofV.size()*size2);
	hipMalloc(&col,(size_t)colV.size()*size2);
	hipMemcpyAsync(rof,&rofV[0],(size_t)rofV.size()*size2,hipMemcpyHostToDevice,stream[4]);
	hipMemcpyAsync(col,&colV[0],(size_t)colV.size()*size2,hipMemcpyHostToDevice,stream[5]);

	int stride = (int)(nm*nm);
	if(timer){
		hipDeviceSynchronize();
		chrono::time_point<std::chrono::system_clock> startg = std::chrono::system_clock::now();
		matMulGPU<<<stride,m*m,2*size*m*m,0>>>(da,db,dc,ka,kb,rof,col,m,n,mp1.size(),mp2.size()); // i X k
		hipDeviceSynchronize();
		chrono::time_point<std::chrono::system_clock> endg = std::chrono::system_clock::now();
		chrono::duration<double> elapsed_secondsg = endg-startg;
		std::cout << "gpu multiplication time: " << elapsed_secondsg.count() << "s\n";
	}else{
		hipDeviceSynchronize();
		matMulGPU<<<stride,m*m,2*size*m*m,0>>>(da,db,dc,ka,kb,rof,col,m,n,mp1.size(),mp2.size()); // i X k
	}

	hipDeviceSynchronize();
	err = hipMemcpy((void *)c,(void *)dc,size3,hipMemcpyDeviceToHost);
	if (err != hipSuccess) 
	    printf("Error: %s\n", hipGetErrorString(err));

	// free the memory
	hipFree(da);
	hipFree(db);
	hipFree(ka);
	hipFree(kb);
	hipFree(rof);
	hipFree(col);
	hipFree(dc);
	for(int i = 0;i<streamSize ;i++){
		hipStreamDestroy(stream[i]);
	}
}

int main(int argc, char const *argv[])
{
	string inputA = "";
	string inputB = "";
	string outputC = "";

	// int thrds = 8;
	// std::ofstream myFile;
	// myFile.open("Mat.txt");
	chrono::time_point<std::chrono::system_clock> start, end;
	chrono::duration<double> elapsed_seconds;
	start = chrono::system_clock::now();
	
	if(argc == 4){
		inputA = argv[1];
		inputB = argv[2];
		outputC = argv[3];
	}else{
		printf("Incorrect number of arguments\n");
		cout << "Given: " << argc - 1 << " Expected: 3" << endl;
		return 0;
	}
	char buffer1[4];
	char buffer2[4];

    std::ifstream fileA;
    fileA.open(inputA,std::ios::binary);
	std::ifstream fileB;
	fileB.open(inputB,std::ios::binary);
    std::ofstream outC;
    outC.open(outputC);


    fileA.read(buffer1,4);
    int n1 = givint(buffer1);
    fileA.read(buffer1,4);
    int m1 = givint(buffer1);
    fileA.read(buffer1,4);
    int k1 = givint(buffer1);

	fileB.read(buffer2,4);
	int n2 = givint(buffer2);
	fileB.read(buffer2,4);
	int m2 = givint(buffer2);
	fileB.read(buffer2,4);
	int k2 = givint(buffer2);

	assert(n1 == n2);
	assert(m1 == m2);
	int n = n1, m = m1;

	vector<array<int,3>> keysa;
	vector<uint> vala;
	vector<array<int,3>> keysb;
	vector<uint> valb;

    // std::map<pair<int,int>,vector<uint>> inMapA;
    for (int i = 0; i < k1; ++i){
	    fileA.read(buffer1,4);
	    int a = givint(buffer1);
	    fileA.read(buffer1,4);
	    int b = givint(buffer1);
		array<int,3> pt = {a,b,i};
		keysa.push_back(pt);
	    vector<uint> vt;
	    for (int j = 0; j < m; ++j){
		  	for (int h = 0; h < m; ++h){
				int e = 0;
			  	fileA.read((char *)&e,2);
			    vt.push_back(e);
		  	}
	    }
		vala.insert(vala.end(),vt.begin(),vt.end());
    }
	fileA.close();
	// std::map<pair<int,int>,vector<uint>> inMapB;
	for (int i = 0; i < k2; ++i)
    {
	    fileB.read(buffer2,4);
	    int a = givint(buffer2);
	    fileB.read(buffer2,4);
	    int b = givint(buffer2);
		array<int,3> pt = {a,b,i};
		keysb.push_back(pt);
	    vector<uint> vt;
	    for (int j = 0; j < m; ++j){
		  	for (int h = 0; h < m; ++h){
				int e = 0;
			  	fileB.read((char *)&e,2);
			    vt.push_back(e);
		  	}
	    }
		valb.insert(valb.end(),vt.begin(),vt.end());
    }
	fileB.close();
    cout << "Input reading done, n: " << n << " m: " << m <<  " k values: " << keysa.size() << " and " << keysb.size() << endl;
	
	sort(keysa.begin(), keysa.end());
	sort(keysb.begin(), keysb.end());
	
	assert(keysa.size() == vala.size()/m/m);
	assert(keysb.size() == valb.size()/m/m);
	assert(keysa.size() == k1);
	assert(keysb.size() == k2);
	// inMapA.clear();
	// inMapB.clear();
	end = chrono::system_clock::now();
	elapsed_seconds = end-start;
	std::cout << "Sorted input: " << elapsed_seconds.count() << endl;

	// vector<pair<int,int>> keysc;
	vector<uint> valc((size_t)n*(size_t)n,0);
	start = chrono::system_clock::now();
	matMul(keysa, vala, keysb, valb, n, m, valc);
	end = chrono::system_clock::now();
	elapsed_seconds = end-start;
	std::cout << "Time taken for matrix multiplication: " << elapsed_seconds.count() << endl;

	// readable output
	int nm = n/m;
    cout << "Writing output" << endl;
    outC.write((char *)&n,4);
    outC.write((char *)&m,4);
    long kout = 0;
    outC.write((char *)&kout,4);
	for(int i = 0; i < nm; ++i)
	{	
		for (int j = 0; j < nm; ++j)
		{
			vector<uint> vtemp(valc.begin()+(i*nm + j)*m*m,valc.begin()+(i*nm + j + 1)*m*m);
			if(!isZero(&vtemp[0],m*m)){
				int aout = i;
				outC.write((char *)&aout,4);
				int bout = j;
				outC.write((char *)&bout,4);
				for (int k = 0; k < m*m; ++k){
					uint eout = vtemp[k];
					outC.write((char *)&eout,4);
				}
				kout += 1;
			}
		}
	}
    cout << "Number of output non-zero blocks: " << kout << endl;
    outC.seekp(8);
    outC.write((char *)&kout,4);
    cout << "Writing done" << endl;
    outC.close();
	// myFile.close();
	return 0;
}