#include "hip/hip_runtime.h"
#include "matrix.h"

int main(int argc, char const *argv[])
{
	string inputA = "";
	string inputB = "";
	string outputC = "";

	// int thrds = 8;
	std::ofstream myFile;
	if(argc == 4){
		inputA = argv[1];
		inputB = argv[2];
		outputC = argv[3];
	}else{
		printf("Incorrect number of arguments\n");
		cout << "Given: " << argc - 1 << " Expected: 3" << endl;
		return 0;
	}
	char buffer1[4];
	char buffer2[4];

    std::ifstream fileA;
    fileA.open(inputA,std::ios::binary);
	std::ifstream fileB;
	fileB.open(inputB,std::ios::binary);
    std::ofstream outC;
    outC.open(outputC);


    fileA.read(buffer1,4);
    int n1 = givint(buffer1);
    fileA.read(buffer1,4);
    int m1 = givint(buffer1);
    fileA.read(buffer1,4);
    int k1 = givint(buffer1);

	fileB.read(buffer2,4);
	int n2 = givint(buffer2);
	fileB.read(buffer2,4);
	int m2 = givint(buffer2);
	fileB.read(buffer2,4);
	int k2 = givint(buffer2);

	assert(n1 == n2);
	assert(m1 == m2);
	int n = n1, m = m1;

    std::map<pair<int,int>,vector<vector<uint>>> inMapA;
    for (int i = 0; i < k1; ++i){
	    fileA.read(buffer1,4);
	    int a = givint(buffer1);
	    fileA.read(buffer1,4);
	    int b = givint(buffer1);
	    pair<int,int> pt = {a,b};
	    vector<vector<uint>> vt;
	    for (int j = 0; j < m; ++j){
	    	vector<uint> vt1;
		  	for (int h = 0; h < m; ++h){
				int e = 0;
			  	fileA.read((char *)&e,2);
			    vt1.push_back(e);
		  	}
		  	vt.push_back(vt1);
	    }
	    inMapA[pt] = vt; 
    }
	fileA.close();
	std::map<pair<int,int>,vector<vector<uint>>> inMapB;
	for (int i = 0; i < k2; ++i)
    {
	    fileB.read(buffer2,4);
	    int a = givint(buffer2);
	    fileB.read(buffer2,4);
	    int b = givint(buffer2);
	    pair<int,int> pt = {a,b};
	    vector<vector<uint>> vt;
	    for (int j = 0; j < m; ++j)
	    {
	    	vector<uint> vt1;
		  	for (int h = 0; h < m; ++h)
		  	{
				int e = 0;
			  	fileB.read((char *)&e,2);
			    vt1.push_back(e);
		  	}
		  	vt.push_back(vt1);
	    }
	    inMapB[pt] = vt; 
    }
	fileB.close();
    cout << "Input reading done, k values: " << inMapA.size() << " and " << inMapB.size() << endl;

    std::map<pair<int,int>,vector<vector<uint>>> outMap;
	matMul(inMapA, inMapB, n, m, k1, k2, outMap);

	// readable output
    myFile.open("Mat.txt");
    printmap(outMap,myFile);
	myFile.close();

    cout << "Writing output" << endl;
    outC.write((char *)&n,4);
    outC.write((char *)&m,4);
    int kout = 0;
    outC.write((char *)&kout,4);
    for(auto i = outMap.begin(); i != outMap.end(); i++){
	    int aout = i->first.first;
	    outC.write((char *)&aout,4);
	    int bout = i->first.second;
	    outC.write((char *)&bout,4);
	    for (int j = 0; j < m; ++j){
		  	for (int h = 0; h < m; ++h){
			  	uint eout = min(i->second[j][h],(uint)MAX_VAL);
			  	outC.write((char *)&eout,4);
		  	}
	    }
	    kout += 1;
    }

    cout << "Number of output non-zero blocks: " << kout << endl;
    outC.seekp(8);
    outC.write((char *)&kout,4);
    cout << "Writing done" << endl;
    outC.close();
	return 0;
}