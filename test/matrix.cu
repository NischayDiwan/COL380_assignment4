#include "hip/hip_runtime.h"
#include "matrix.h"

void printvec(vector<uint> &a,ofstream &outstr){
   	for(int i=0; i < a.size(); i++)
		outstr << a.at(i) << " ";
	outstr << endl;
}

void printmap(map<pair<int,int>,vector<vector<uint>>> &m, ofstream &outstr){
	for(auto i = m.begin(); i != m.end(); i++){
		outstr << i->first.first << "," << i->first.second << ":\n";
		for (int j = 0; j < (i->second).size(); ++j)
		{
			printvec((i->second)[j],outstr);
		}
	}
}

void inline transpose(vector<vector<uint>> &a,vector<vector<uint>> &b){
	for(int i=0; i < a.size(); i++){
		for(int j=0; j < a.size(); j++){
			b[i][j] = a[j][i];
		}
	}
}

int givint(char *buffer){
	int *res = (int *)(buffer);
	return *res;
}

void inline blockOuter(vector<vector<uint>> &v1,vector<vector<uint>> &v2, vector<vector<uint>> &r){
	for (int i = 0; i < v1.size(); ++i)
	{	
		for (int j = 0; j < v1.size(); ++j)
		{
			r[i][j] = v1[i][j] + v2[i][j];
		}
	}
}

void inline blockInner(vector<vector<uint>> &v1,vector<vector<uint>> &v2, vector<vector<uint>> &r){
	for (int i = 0; i < v1.size(); ++i)
	{	
		for (int j = 0; j < v1.size(); ++j)
		{
			uint temp = 0;
			for (int k = 0; k < v1.size(); ++k)
			{
				temp = temp + (v1[i][k] * v2[k][j]);
			}
			r[i][j] = temp;
		}
	}
}

bool inline isZero(vector<vector<uint>> &v){
	bool flg = true;
	for (int i = 0; i < v.size(); ++i)
	{	
		if(flg == false){
			break;
		}
		for (int j = 0; j < v.size(); ++j)
		{
			if(v[i][j] != 0){
				flg = false;
				break;
			}
		}
	}
	return flg;
}

// __global__
// void matMulGPU(void){
// 	int bid = blockIdx.x;
// 	int tid = threadIdx.x;
// 	int gtid = bid*blockDim.x + tid;
// }

void matMul(map<pair<int,int>,vector<vector<uint>>> &mp1, map<pair<int,int>,vector<vector<uint>>> &mp2, int n, int m, int k1, int k2, map<pair<int,int>,vector<vector<uint>>> &resm){
	int nm = n/m;
	// matMulGPU<<<m,m>>>();

	vector<vector<vector<uint>>> valV;
	vector<uint> colV;
	vector<uint> rofV;
	// vector<uint> rowV;
	int offset = 0;
	int rowno = 0;
	rofV.push_back(offset);
	for(auto i = mp1.begin(); i != mp1.end(); i++){
		valV.push_back(i->second);
		colV.push_back(i->first.second);
		// rowV.push_back(i->first.first);
		if(i->first.first > rowno){
			for(int cc = 0;cc<(i->first.first-rowno);cc++){
				rofV.push_back(offset);
			}
			rowno = i->first.first;
		}
		offset+=1;
	}
	for(int j = rowno;j<nm;j++){
		rofV.push_back(valV.size());
	}	
	for(int i = 0; i < nm; ++i)
	{	
		for (int k = 0; k < nm; ++k)
		{
			vector<vector<uint>> vtemp(m,vector<uint>(m,0));
			for (int j = rofV[i]; j < rofV[i+1]; j++)
			{
				int cl = colV[j];
				if(!(mp2.find({cl,k}) == mp2.end())){
					vector<vector<uint>> vtemp1(m,vector<uint>(m,0));
					blockInner(valV[j],mp2[{cl,k}],vtemp1);
					blockOuter(vtemp,vtemp1,vtemp);
				}
			}
			if(!isZero(vtemp)){
				pair<int,int> ptemp = {i,k};
				resm[ptemp] = vtemp;
			}
		}
	}
}
